#include "hip/hip_runtime.h"
#include "lbvh.cuh"
#include <random>
#include <vector>
#include <thrust/random.h>
#include <torch/extension.h>

struct bounding_box {
    float x_min;
    float x_max;
    float y_min;
    float y_max;
    float z_min;
    float z_max;
};

struct aabb_getter
{
    __device__
    lbvh::aabb<float> operator()(const struct bounding_box f) const noexcept
    {
        lbvh::aabb<float> retval;
        retval.upper.x = f.x_max;
        retval.upper.y = f.y_max;
        retval.upper.z = f.z_max;
        retval.lower.x = f.x_min;
        retval.lower.y = f.y_min;
        retval.lower.z = f.z_min;
        return retval;
    }
};
// struct distance_calculator
// {
//     __device__
//     float operator()(const float4 point, const float4 object) const noexcept
//     {
//         return (point.x - object.x) * (point.x - object.x) +
//                (point.y - object.y) * (point.y - object.y) +
//                (point.z - object.z) * (point.z - object.z);
//     }
// };

std::tuple<torch::Tensor, torch::Tensor>
BuildBVH (
    const torch::Tensor& objects // Shape (N, 3, 2)
) {
    size_t N = objects.size(0);
    auto int_opts = objects.options().dtype(torch::kInt32);
    auto float_opts = objects.options().dtype(torch::kFloat32);

    bounding_box *object_bbs = (bounding_box *)objects.contiguous().data<float>();
    std::vector<bounding_box> objects_vec(object_bbs, object_bbs + N);

    lbvh::bvh<float, bounding_box, aabb_getter> bvh(objects_vec.begin(), objects_vec.end(), true);

    auto nodes = bvh.nodes_host();
    auto aabbs = bvh.aabbs_host();

    torch::Tensor bvh_nodes = torch::from_blob(nodes.data(), {(long)nodes.size(), 4}, torch::kInt32).clone();
    torch::Tensor bvh_aabbs = torch::from_blob(aabbs.data(), {(long)aabbs.size(), 2, 4}, torch::kFloat32).clone();
    bvh_aabbs = bvh_aabbs.narrow(2, 0, 3);
    bvh_aabbs = bvh_aabbs.permute({0, 2, 1});

    return std::make_tuple(bvh_nodes, bvh_aabbs);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("BuildBVH", &BuildBVH, "Build BVH (CUDA)");
}
